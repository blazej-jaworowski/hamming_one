#include "hip/hip_runtime.h"
#include "HOGPU.cuh"

#include <cstdio>

void HOGPU::load_data(const uint8_t *data, int l, int n) {
    dword_length = (l - 1) / 32 + 1;
    this->n = n;

    uint32_t *host_data = new uint32_t[n * dword_length]();

    int bit_index = 0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < l; j++) {
            bool bit = ((data[bit_index / 8] >> (7 - bit_index % 8))) & 1;

            host_data[i * dword_length + j / 32] |= bit << (31 - j % 32);
            bit_index++;
        }
    }

    hipMalloc(&this->data, n * dword_length * 4);
    hipMemcpy(this->data, host_data, n * dword_length * 4,
               hipMemcpyHostToDevice);
}

__global__ void print_pairs_kernel(uint32_t *data, int n, int dword_length) {
    int i = blockIdx.x * 1024 + threadIdx.x;
    if (i >= n)
        return;
    for (int j = 0; j < i; j++) {
        bool distance_one = false;
        bool differs = false;
        for (int k = 0; k < dword_length; k++) {
            uint32_t prod =
                data[i * dword_length + k] ^ data[j * dword_length + k];
            if (prod == 0)
                continue;
            if ((prod & (prod - 1)) == 0) {
                if (differs) {
                    distance_one = false;
                    break;
                }
                differs = true;
                distance_one = true;
                continue;
            }
            distance_one = false;
            break;
        }

        if (distance_one) {
            printf("%d %d\n", j, i);
        }
    }
}

void HOGPU::print_pairs() {
    print_pairs_kernel<<<(n - 1) / 1024 + 1, 1024>>>(data, n, dword_length);
    hipDeviceSynchronize();
};

HOGPU::~HOGPU() { hipFree(data); }